#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------------------------------------------------------
/*
Nanowire simulation parallel code implemented using CUDA API
Author: Abhinav Roy
Date: 22/01/2021
*/
//-------------------------------------------------------------------------------------------------------------------------

// Including all the required header files
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
//-------------------------------------------------------------------------------------------------------------------------
// Including all the supplementary code
// #include "ran2.c"		//Pseudo random number generator code
#include "real2complex.cu"
#include "complex2real.cu"
#include "complex2real_y.cu"
#include "solve_cahn_hilliard.cu"
#include "compute_variable_mobility.cu"
#include "compute_g.cu"
#include "compute_q.cu"
#include "compute_y.cu"
#include "compute_r.cu"
//-------------------------------------------------------------------------------------------------------------------------
// Defining the value of PI upto 15 decimal places

#define PI 3.141592653589793

//-------------------------------------------------------------------------------------------------------------------------

 #define RESTART

//-------------------------------------------------------------------------------------------------------------------------
#include "hip/hip_runtime_api.h"
#include <unistd.h>
//#include <complex.h>
/*
// Header files for the cuRAND library
#include <hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
*/

#include "gsl/gsl_math.h"
#include "gsl/gsl_rng.h"


//-------------------------------------------------------------------------------------------------------------------------
int main(void)
{

	//-------------------------------------------------------------------------------------------------------------------------
	// Declaration of the variables
	FILE *fp, *fr;
	char file_name[100];
	int i1, i2, i3;
	double dt, dx, dy, dz, M, kappa, A;
	double delkx, delky, delkz, kx, ky, kz;
	int Nx, Ny, Nz, halfNx, halfNy, halfNz;
	int temp, time_step, start_time, end_time;
	double time_elapsed = 0.0;
	time_t begin_t, end_t;
	int block_size_x, block_size_y;
	hipError_t err;
	// Saving the start time of the simulation run
	begin_t = time(NULL);
	//-------------------------------------------------------------------------------------------------------------------------

	/*
	// cuRAND pseudo random number generator

	const unsigned int DEFAULT_SEED = 777;
	int rand_n = 1;
	float *d_Rand;
	checkCudaErrors(hipMalloc((void **)&d_Rand, rand_n * sizeof(float)));
	// parsing the seed
    	int seed = DEFAULT_SEED;

    	hipStream_t stream;
	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	hiprandGenerator_t prngGPU;
   	checkCudaErrors(hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32));
   	checkCudaErrors(hiprandSetStream(prngGPU, stream));
   	checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed));
	*/

	//-------------------------------------------------------------------------------------------------------------------------

	// Variables initialization for the simulation
	if ((fr = fopen("./time_data.dat","r")) == NULL)
	{
		printf("Unable to open the time data input file. Exiting!\n");
	}
	else
	{
		fr = fopen("./time_data.dat","r");
	}
	(void) fscanf(fr,"%d%d%d", &start_time, &end_time, &time_step);
	(void) fclose(fr);

	// simulation system parametrs
	Nx = 768;
	Ny = 768;
	Nz = 96;
	dx = dy = dz = 0.5;

	// time step value
	dt = 1.0;

	// half of the simulation system size
	halfNx = (int) Nx/2;
	halfNy = (int) Ny/2;
	halfNz = (int) Nz/2;


	// for defining the Fourier modes
	delkx = 2*PI/(Nx*dx);
	delky = 2*PI/(Ny*dy);
	delkz = 2*PI/(Nz*dz);

	// Simulation parameters
	M = 1.0;
	A = 1.0;
	kappa = 1.0;

	// Stability factor
	double alpha = 0.5;

	// Defining the number of threads in the x and y dimensions of a block (maximum 32*32 = 1024 threads per block allowed)
	block_size_x = 32;
	block_size_y = 32;

	//-------------------------------------------------------------------------------------------------------------------------
	dim3 dimBlock(block_size_x, block_size_y , 1);
	dim3 dimGrid (256, 216);
	//-------------------------------------------------------------------------------------------------------------------------
	// I guess this is to make the numbers x and y the next whole numbers (From Arijit)
	if ( Nx%block_size_x != 0)
	{
		dimGrid.x += 1;
	}
	if (Ny%block_size_y!=0)
	{
		dimGrid.y += 1;
	}
	//-------------------------------------------------------------------------------------------------------------------------
	//checking whether there are sufficient no of blocks
	printf("\nThe no. of blocks created in the x-direction=%d\n",dimGrid.x);
	printf("The no. of blocks created in the y-direction=%d\n",dimGrid.y);
	printf("The no. of blocks created in the z-direction=%d\n",dimGrid.z);
	//-------------------------------------------------------------------------------------------------------------------------
	// Declaring all the host variables
	double *c_host, *k2_vect_host, *kx_vect_host, *ky_vect_host, *kz_vect_host;
	// Declaring arrays for the device
	hipfftComplex *c_device_f, *g_device_f, *r_device_f, *y1_device_f, *y2_device_f, *y3_device_f, *qx_device_f, *qy_device_f, *qz_device_f;
	double *c_device, *g_device, *phi_device, *y1_device, *y2_device, *y3_device, *k2_vect_device, *kx_vect_device, *ky_vect_device, *kz_vect_device;
	//-------------------------------------------------------------------------------------------------------------------------

	// Allocating memory to the arrays in the host
	c_host = (double *) malloc(Nx*Ny*Nz*sizeof(double));
	k2_vect_host = (double *) malloc(Nx*Ny*Nz*sizeof(double));
	kx_vect_host = (double *) malloc(Nx*Ny*Nz*sizeof(double));
	ky_vect_host = (double *) malloc(Nx*Ny*Nz*sizeof(double));
	kz_vect_host = (double *) malloc(Nx*Ny*Nz*sizeof(double));

	// Allocating memory to the arrays in the device
	hipMalloc((void**)&c_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&g_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&k2_vect_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&kx_vect_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&ky_vect_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&kz_vect_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&y1_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&y2_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&y3_device, sizeof(double)*Nx*Ny*Nz);
	hipMalloc((void**)&phi_device, sizeof(double)*Nx*Ny*Nz);
	// hipfftComplex type variables
	hipMalloc((void**)&c_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&g_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&r_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&y1_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&y2_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&y3_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&qx_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&qy_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);
	hipMalloc((void**)&qz_device_f, sizeof(hipfftComplex)*Nx*Ny*Nz);

	//-------------------------------------------------------------------------------------------------------------------------
	// Defining the Fourier modes

	for (i1 = 0 ; i1 < Nx; ++i1)
	{
		if(i1 <= halfNx)
		{
			kx = i1*delkx;
		}
		else
		{
			kx = (i1-Nx)*delkx;
		}
		for(i2 = 0; i2 < Ny; ++i2)
		{
			if(i2 <= halfNy)
			{
				ky = i2*delky;
			}
			else
			{
				ky = (i2-Ny)*delky;
			}
			for (i3 = 0; i3 < Nz; ++i3)
			{
				if(i3 <= halfNz)
				{
					kz = i3*delkz;
				}
				else
				{
					kz = (i3-Nz)*delkz;
				}
				k2_vect_host[i3 + Nz*(i2 + Ny*i1)] = kx*kx + ky*ky + kz*kz;
				kx_vect_host[i3 + Nz*(i2 + Ny*i1)] = kx;
				ky_vect_host[i3 + Nz*(i2 + Ny*i1)] = ky;
				kz_vect_host[i3 + Nz*(i2 + Ny*i1)] = kz;
			}

		}
	}

	// Transfer the k2, kx, ky vector to device
	hipMemcpy(k2_vect_device, k2_vect_host, Nx*Ny*Nz*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(kx_vect_device, kx_vect_host, Nx*Ny*Nz*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(ky_vect_device, ky_vect_host, Nx*Ny*Nz*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(kz_vect_device, kz_vect_host, Nx*Ny*Nz*sizeof(double), hipMemcpyHostToDevice);

	err = hipPeekAtLastError();
	if (err != hipSuccess)
	{
		printf("Error 1: %s\n", hipGetErrorString(err));
	}
	//-------------------------------------------------------------------------------------------------------------------------
							/*RESTART OF SIMULATION*/
	//-------------------------------------------------------------------------------------------------------------------------
	#ifdef RESTART
	{
		printf("\nCode execution for the restarted simulation has commenced:\n");
		sprintf(file_name, "./output/time%d.dat", start_time);
		if ((fp = fopen(file_name,"rb")) == NULL)
		{
			printf("Unable to open data file. Exiting.\n");
			exit(0);
		}
		else
		{
			fp = fopen(file_name,"rb");
		}
		(void) fread(&c_host[0], sizeof(double),(size_t) Nx*Ny*Nz, fp);
		(void) fclose (fp);
		fflush(fp);

	}

	//-------------------------------------------------------------------------------------------------------------------------
							/*NEW SIMULATION*/
	//-------------------------------------------------------------------------------------------------------------------------
	#else
	{
		printf("\nCode execution for a new simulation has started:\n");

		//Remove all the existing output]
		(void) system("rm -rf ./output/*");
		//-------------------------------------------------------------------------------------------------------------------------
		unsigned long int seed_val = 5454;
		// long SEED = 654321;
		//-------------------------------------------------------------------------------------------------------------------------
		double R1, R2;
		double c_zero, c_noise;

		// Radius of the nanowires
		R1 = 12.0;
		R2 = 14.0;

		// The composition/density variable

		c_zero = 1.0;
		c_noise = 1.0e-03;

		//Setting the initial composition profile.
		for (i1 = 0; i1 < Nx; ++i1)
		{
			for (i2 = 0; i2 < Ny; ++i2)
			{
				for (i3 = 0; i3 < Nz; ++i3)
				{
					c_host[i3 + Nz*(i2 + Ny*i1)] = 0.0;
				}

			}
		}
		//-------------------------------------------------------------------------------------------------------------------------

		// ORIENTATION OF 30 DEGREES BETWEEN THE RODS
		/*
		int *center1, *center2;
		center1 = (int *) malloc((size_t) Nz*sizeof(int));
		center2 = (int *) malloc((size_t) Nz*sizeof(int));
		int C1, C2;
		C1 = halfNx;
	        C2 = halfNx + R1 + R2;

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - C1)*(i1 - C1) + (i2 - halfNy)*(i2 - halfNy) < R1*R1)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

	        for (i2=0; i2 < Ny; ++i2)
	        {
	                for (i3=0; i3 < Nz; ++i3)
	                {
	                        if (fabs((i3 - halfNz) - (sqrt(3))*(i2 - halfNy)) <= 2/sqrt(3))
	                        {
	                                center1[i3] = i3;
	                                center2[i3] = i2;
	                        }
	                }
	        }

	        for (i1 = 0; i1 < Nx; ++i1)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
						if(i3 > 30 && i3 < Nz - 30 && i2 > 30 && i2 < Ny - 30)
						{

	                                                        //if ((i3 - center1[i3])*(i3 - center1[i3]) + (i2 - center2[i3])*(i2 - center2[i3]) + (i1 - C2)*(i1 - C2)< R2*R2)
								if (fabs((i2 - center2[i3])*(i2 - center2[i3])*3/4 + (i1 - C2)*(i1 - C2))< R2*R2)
	                                                      {
	                                                                c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                                      }

						}

	                        }
			}
	        }
		// Free all the dynamically allocated variables for storing the central axis of different wire orientations
		free(center1);
		free(center2);
		*/
		//-------------------------------------------------------------------------------------------------------------------------

		// ORIENTATION OF 60 DEGREES BETWEEN THE RODS
		/*
		int *center1, *center2;
		center1 = (int *) malloc((size_t) Nz*sizeof(int));
		center2 = (int *) malloc((size_t) Nz*sizeof(int));
		int  C1,C2;
		C1 = halfNx;
		C2 = halfNx + R1 + R2;

		//Setting the initial density profile.
		for (i1=0; i1<Nx; i1++)
		{
		        for (i2=0; i2<Ny; i2++)
		        {
		                for (i3=0; i3<Nz; i3++)
		                {
		                        c_host[i3 + Nz*(i2 + Ny*i1)] = 0.0;
		                }
		        }
		}

		for(i1=0; i1 < Nx; i1++)
		{
		        for(i2=0; i2 < Ny; i2++)
		        {
		                for (i3=0; i3<Nz; i3++)
		                {
		                        if ((i1 - C1)*(i1 - C1) + (i2 - halfNy)*(i2 - halfNy) < R1*R1)
		                        {
		                                c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

		                        }
		                }
		        }
		}

		for (i2=0; i2 < Ny; ++i2)
		{
		        for (i3=0; i3 < Nz; ++i3)
		        {
		                if (fabs((i3 - halfNz) - (1/sqrt(3))*(i2 - halfNy)) <= (13*sqrt(3))/2)
		                {
		                        center1[i3] = i3;
		                        center2[i3] = i2;
		                }
		        }
		}

		for (i1 = 0; i1 < Nx; ++i1)
		{
		        for(i2=0; i2 < Ny; i2++)
		        {
		                for (i3=0; i3<Nz; i3++)
		                {
						if(i3 > 30 && i3 < Nz - 30 && i2 > 30 && i2 < Ny - 30)
						{

		                                                //if (round(pow(sin(M_PI/3),2)*(i3 - center1[i3])*(i3 - center1[i3]))+ round(pow(sin(M_PI/3),2)*(i2 - center2[i3])*(i2- center2[i3])) + (i1 - C2)*(i1 - C2) <= R2*R2)

								if (fabs((i2 - center2[i3])*(i2 - center2[i3])*1/4 + (i1 - C2)*(i1 - C2))< R2*R2)
								{
		                                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;


		                                              }




						}

		                }
			}
		}
		// Free all the dynamically allocated variables for storing the central axis of different wire orientations
		free(center1);
		free(center2);
		*/
		//-------------------------------------------------------------------------------------------------------------------------

		// MULTIPLE JUNCTIONS
		/*
		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - halfNx)*(i1 - halfNx) + (i2 - halfNy - halfNy/2)*(i2 - halfNy - halfNy/2) < R1*R1)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - halfNx)*(i1 - halfNx) + (i2 - halfNy)*(i2 - halfNy) < R1*R1)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - halfNx)*(i1 - halfNx) + (i2 - halfNy + halfNy/2)*(i2 - halfNy + halfNy/2) < R1*R1)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - (halfNx + R1 + R2))*(i1 - (halfNx + R1 + R2)) + (i3 - halfNz - halfNz/2)*(i3 - halfNz - halfNz/2) < R2*R2)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - (halfNx + R1 + R2))*(i1 - (halfNx + R1 + R2)) + (i3 - halfNz)*(i3 - halfNz) < R2*R2)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - (halfNx + R1 + R2))*(i1 - (halfNx + R1 + R2)) + (i3 - halfNz + halfNz/2)*(i3 - halfNz + halfNz/2) < R2*R2)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }
		*/

		//-------------------------------------------------------------------------------------------------------------------------
		// ORIENTATION OF 45 DEGREES BETWEEN THE RODS
		/*
		int *center1, *center2;
		center1 = (int *) malloc((size_t) Nz*sizeof(int));
		center2 = (int *) malloc((size_t) Nz*sizeof(int));
		int C1, C2;
		C1 = halfNx;
		C2 = halfNx + R1 + R2;

		for(i1=0; i1 < Nx; i1++)
		{
			for(i2=0; i2 < Ny; i2++)
			{
				for (i3=0; i3<Nz; i3++)
				{
					if ((i1 - C1)*(i1 - C1) + (i2 - halfNy)*(i2 - halfNy) < R1*R1)
					{
						c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

					}
				}
			}
		}

		for (i2=0; i2 < Ny; ++i2)
		{
			for (i3=0; i3 < Nz; ++i3)
			{
				if (i2 == i3)
				{
					center1[i3] = i3;
					center2[i3] = i2;
				}
			}
		}

		for (i1 = 0; i1 < Nx; ++i1)
		{
			for(i2=0; i2 < Ny; i2++)
			{
				for (i3=0; i3<Nz; i3++)
				{
						if(i3 > 30 && i3 < Nz - 30 && i2 > 30 && i2 < Ny - 30)
						{


								//if ((i3 - center1[i3])*(i3 - center1[i3]) + (i2 - center2[i3])*(i2 - center2[i3]) + (i1 - C2)*(i1 - C2)< R2*R2)
								if (fabs((i2 - center2[i3])*(i2 - center2[i3])/2 + (i1 - C2)*(i1 - C2))< R2*R2)
								{
									c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;
								}

						}

				}
			}
		}
		// Free all the dynamically allocated variables for storing the central axis of different wire orientations
		free(center1);
		free(center2);
		*/

		//-------------------------------------------------------------------------------------------------------------------------

		// ORIENTATION OF 90 DEGREES BETWEEN THE RODS
		/**/
	        for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - halfNx)*(i1 - halfNx) + (i2 - halfNy)*(i2 - halfNy) < R1*R1)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

	        for(i1=0; i1 < Nx; i1++)
	        {
	                for(i2=0; i2 < Ny; i2++)
	                {
	                        for (i3=0; i3<Nz; i3++)
	                        {
	                                if ((i1 - (halfNx + R1 + R2))*(i1 - (halfNx + R1 + R2)) + (i3 - halfNz)*(i3 - halfNz) < R2*R2)
	                                {
	                                        c_host[i3 + Nz*(i2 + Ny*i1)] = c_zero;

	                                }
	                        }
	                }
	        }

		//-------------------------------------------------------------------------------------------------------------------------
		// Introducing noise in the density variable

		// GSL Tausworthe random number generator.
		gsl_rng * ran_num;
		const gsl_rng_type * Taus;
		Taus = gsl_rng_taus2;
		ran_num = gsl_rng_alloc(Taus);
		gsl_rng_set(ran_num,seed_val);

		for (i1 = 0; i1 < Nx; ++i1)
		{
			for (i2 = 0; i2 < Ny; ++i2)
			{
				for (i3 = 0; i3 < Nz; ++i3)
				{
					// c_host[i3 + Nz*(i2 + Ny*i1)] += c_noise*(2.0*ran2(&SEED) - 1.0);
					c_host[i3 + Nx*(i2+Ny*i1)] += c_noise*(0.5 - gsl_rng_uniform_pos(ran_num));
				}

			}
		}
		//-------------------------------------------------------------------------------------------------------------------------
		// Writing the initial composition profile

		sprintf(file_name,"./output/time%d.dat", start_time);
		fp = fopen(file_name,"wb");
		fwrite(&c_host[0], sizeof(double),(size_t) Nx*Ny*Nz, fp);
		(void) fclose(fp);
		fflush(fp);

	}
	#endif
	//-------------------------------------------------------------------------------------------------------------------------
	// Define fft plan for CUFFT
	hipfftHandle plan;
	hipfftPlan3d(&plan, Nx, Ny, Nz, HIPFFT_C2C);

	// Copy the initial configuration from the host to the device

	hipMemcpy(c_device, c_host, sizeof(double)*Nx*Ny*Nz, hipMemcpyHostToDevice);
	err = hipPeekAtLastError();
	if (err != hipSuccess)
	{
		printf("Error 3: %s\n", hipGetErrorString(err));
	}
	//-------------------------------------------------------------------------------------------------------------------------
	// 						Temporal evolution loop
	//-------------------------------------------------------------------------------------------------------------------------
	for (temp = start_time + 1; temp < end_time + 1; ++temp)
	{
		//-----------------------------------------------------------------------------------------------
		// Calculate the value of g
		compute_g <<<dimGrid, dimBlock>>> (c_device, g_device, A, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 4: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		compute_variable_mobility <<<dimGrid, dimBlock>>> (phi_device, c_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 5: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		// Move composition and g from real to complex on the device
		real2complex <<<dimGrid, dimBlock>>>(c_device_f, c_device, g_device_f, g_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 6: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		// Taking the variables comp and g from real to fourier space.
		hipfftExecC2C(plan, c_device_f, c_device_f, HIPFFT_FORWARD);
		hipfftExecC2C(plan, g_device_f, g_device_f, HIPFFT_FORWARD);
		compute_r <<<dimGrid, dimBlock>>> (r_device_f, c_device_f, g_device_f, kappa, k2_vect_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 7: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		compute_y <<<dimGrid, dimBlock>>> (y1_device_f, y2_device_f, y3_device_f, r_device_f, kx_vect_device, ky_vect_device, kz_vect_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 8: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		hipfftExecC2C(plan, y1_device_f, y1_device_f, HIPFFT_BACKWARD);
		hipfftExecC2C(plan, y2_device_f, y2_device_f, HIPFFT_BACKWARD);
		hipfftExecC2C(plan, y3_device_f, y3_device_f, HIPFFT_BACKWARD);
		//-----------------------------------------------------------------------------------------------
		complex2real_y <<<dimGrid, dimBlock>>> (y1_device_f, y1_device, y2_device_f, y2_device, y3_device_f, y3_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 9: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		compute_q <<<dimGrid, dimBlock>>> (qx_device_f, qy_device_f, qz_device_f, phi_device, y1_device, y2_device, y3_device, M, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 10: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		hipfftExecC2C(plan, qx_device_f, qx_device_f, HIPFFT_FORWARD);
		hipfftExecC2C(plan, qy_device_f, qy_device_f, HIPFFT_FORWARD);
		hipfftExecC2C(plan, qz_device_f, qz_device_f, HIPFFT_FORWARD);
		//-----------------------------------------------------------------------------------------------
		// Solve the variable mobility cahn hilliard equation
		solve_cahn_hilliard <<<dimGrid, dimBlock>>>(c_device_f, qx_device_f, qy_device_f, qz_device_f, k2_vect_device, kx_vect_device, ky_vect_device, kz_vect_device, Nx, Ny, Nz, kappa, alpha, dt);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 11: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		// Bring composition back to real space
		hipfftExecC2C(plan, c_device_f, c_device_f, HIPFFT_BACKWARD);
		//-----------------------------------------------------------------------------------------------
		// Complex to real of both c and g
		complex2real <<<dimGrid, dimBlock>>> (c_device_f, c_device, Nx, Ny, Nz);
		hipDeviceSynchronize();
		err = hipPeekAtLastError();
		if (err != hipSuccess)
		{
			printf("Error 12: %s\n", hipGetErrorString(err));
		}
		//-----------------------------------------------------------------------------------------------
		if(temp%time_step == 0)
		{
			// Copy the composition value from the device to the host for writing
			hipMemcpy(c_host, c_device, sizeof(double)*Nx*Ny*Nz, hipMemcpyDeviceToHost);

			// Change the directory path to "./output/time%d.dat" for linux system
			sprintf(file_name, "./output/time%d.dat", temp);

			//-----------------------------------------------------------------------------------------------
			fp = fopen(file_name,"wb");
			fwrite(&c_host[0], sizeof(double),(size_t) Nx*Ny*Nz, fp);
			(void) fclose (fp);
			fflush(fp);

		}

	}
	//-------------------------------------------------------------------------------------------------------------------------
	// 						Temporal evolution loop ends
	//-------------------------------------------------------------------------------------------------------------------------

	hipfftDestroy(plan);

	// Free all the dynamically allocated variables on the host
	free(c_host);
	free(k2_vect_host);
	free(kx_vect_host);
	free(ky_vect_host);
	free(kz_vect_host);


	// Free all the dynamically allocated variables on the device

	hipFree(c_device);
	hipFree(g_device);
	hipFree(phi_device);
	hipFree(y1_device);
	hipFree(y2_device);
	hipFree(y3_device);
	hipFree(k2_vect_device);
	hipFree(kx_vect_device);
	hipFree(ky_vect_device);
	hipFree(kz_vect_device);
	hipFree(c_device_f);
	hipFree(g_device_f);
	hipFree(r_device_f);
	hipFree(y1_device_f);
	hipFree(y2_device_f);
	hipFree(y3_device_f);
	hipFree(qx_device_f);
	hipFree(qy_device_f);
	hipFree(qz_device_f);

	printf("\nCode execution has completed.\n");

	// Calculation of the total simulation time required
	end_t = time(NULL);
	time_elapsed = (double) (end_t - begin_t);
	printf("\nThe total simulation time elapsed = %f\n", time_elapsed);

	return 0;
}
//-------------------------------------------------------------------------------------------------------------------------
//							END OF CODE
//-------------------------------------------------------------------------------------------------------------------------
